#include "hip/hip_runtime.h"
/*
    To use this source, cite the paper with the following bibtex:

    @inproceedings{DBLP:conf/crypto/SongLG17,
    author    = {Ling Song and Guohong Liao and Jian Guo},
    title     = {{Non-full Sbox Linearization: Applications to Collision Attacks on Round-Reduced Keccak}},
    booktitle = {Advances in Cryptology - {CRYPTO} 2017 - 37th Annual International Cryptology Conference, Santa Barbara, CA, USA, August 20-24, 2017, Proceedings, Part {II}},
    pages     = {428--451},
    year      = {2017},
    crossref  = {DBLP:conf/crypto/2017-2},
    url       = {https://doi.org/10.1007/978-3-319-63715-0_15},
    doi       = {10.1007/978-3-319-63715-0_15},
    timestamp = {Tue, 15 Aug 2017 07:01:19 +0200},
    biburl    = {http://dblp.org/rec/bib/conf/crypto/SongLG17},
    bibsource = {dblp computer science bibliography, http://dblp.org}
    }
    @proceedings{DBLP:conf/crypto/2017-2,
    editor    = {Jonathan Katz and Hovav Shacham},
    title     = {Advances in Cryptology - {CRYPTO} 2017 - 37th Annual International Cryptology Conference, Santa Barbara, CA, USA, August 20-24, 2017, Proceedings, Part {II}},
    series    = {Lecture Notes in Computer Science},
    volume    = {10402},
    publisher = {Springer},
    year      = {2017},
    url       = {https://doi.org/10.1007/978-3-319-63715-0},
    doi       = {10.1007/978-3-319-63715-0},
    isbn      = {978-3-319-63714-3},
    timestamp = {Mon, 14 Aug 2017 14:37:57 +0200},
    biburl    = {http://dblp.org/rec/bib/conf/crypto/2017-2},
    bibsource = {dblp computer science bibliography, http://dblp.org}
    }

    rewritten for mpunks @bxxd
*/

#include "kernel.h"

using namespace std;

void logger(const char *priority, const char *format, va_list ap)
{

    // Sanity-check parameters
    if (!format)
        return;

    va_list ac;
    va_copy(ac, ap);

    struct tm t;
    time_t ltime = time(NULL);
    gmtime_r(&ltime, &t);
    printf("[%04d-%02d-%02d %02d:%02d:%02d] [%s] ",
           (t.tm_year + 1900), (t.tm_mon + 1), t.tm_mday,
           t.tm_hour, t.tm_min, t.tm_sec,
           priority);
    vprintf(format, ac);

    va_end(ac);
}

void log_sensitive(const char *format, ...)
{
    // print_datetime();
#if FULL
    va_list ap;
    va_start(ap, format);
    logger("INFO", format, ap);
    va_end(ap);
#endif
}

void log_info(const char *format, ...)
{
    // print_datetime();

    va_list ap;
    va_start(ap, format);
    logger("INFO", format, ap);
    va_end(ap);
}

void log_err(const char *format, ...)
{
    // print_datetime();

    va_list ap;
    va_start(ap, format);
    logger("ERROR", format, ap);
    va_end(ap);
}

__device__ uint64_t device_difficulty_upper = 0;
__device__ uint64_t device_difficulty_lower = 5731203885580;

__device__ uint64_t device_minor_upper = 0;
__device__ uint64_t device_minor_lower = 0;

texture<unsigned int, 1, hipReadModeElementType>
    texreference_input;

__constant__ uint64_t RC[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808A,
    0x8000000080008000, 0x000000000000808B, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008A,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000A,
    0x000000008000808B, 0x800000000000008B, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
    0x000000000000800A, 0x800000008000000A, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008};

#define ROL(x, n) (((x) << (n)) | ((x) >> ((uint64_t)64 - (n))))

uint64_t rand_uint64(void)
{
    uint64_t r = 0;
    for (int i = 0; i < 64; i += 15 /*30*/)
    {
        r = r * ((uint64_t)RAND_MAX + 1) + rand();
    }
    return r;
}

//assume each inputs have the same input length

__device__ uint32_t device_hash_count = 0;
__device__ uint64_t device_found_nonce = 0;
__device__ uint64_t device_found_minor = 0;

__global__ void Keccak1600(const int inputByte, uint8_t *output, const int outputByte, uint64_t startNonce)
{

    uint32_t num_keccak_blocks = inputByte / (DATA_BLOCK_SIZE << 1);

    uint64_t state00 = 0, state01 = 0, state02 = 0, state03 = 0, state04 = 0,
             state10 = 0, state11 = 0, state12 = 0, state13 = 0, state14 = 0,
             state20 = 0, state21 = 0, state22 = 0, state23 = 0, state24 = 0,
             state30 = 0, state31 = 0, state32 = 0, state33 = 0, state34 = 0,
             state40 = 0, state41 = 0, state42 = 0, state43 = 0, state44 = 0;
    uint64_t tmpState00 = 0, tmpState01 = 0, tmpState02 = 0, tmpState03 = 0, tmpState04 = 0,
             tmpState10 = 0, tmpState11 = 0, tmpState12 = 0, tmpState13 = 0, tmpState14 = 0,
             tmpState20 = 0, tmpState21 = 0, tmpState22 = 0, tmpState23 = 0, tmpState24 = 0,
             tmpState30 = 0, tmpState31 = 0, tmpState32 = 0, tmpState33 = 0, tmpState34 = 0,
             tmpState40 = 0, tmpState41 = 0, tmpState42 = 0, tmpState43 = 0, tmpState44 = 0;
    uint64_t Csum0, Csum1, Csum2, Csum3, Csum4, D0, D1, D2, D3, D4;

    uint64_t thread = blockDim.x * blockIdx.x + threadIdx.x;
    uint64_t nonce = startNonce + thread;

    // nonce = startNonce + device_hash_count;

#if DEBUG
    // printf("nonce=%lu/0x%016x\n", nonce, nonce);
    printf("n=%lu t=%lu nk=%d bdim=%d bid=%d tid=%d\n", nonce, thread, num_keccak_blocks,
           blockDim.x, blockIdx.x, threadIdx.x);

    printf("minor difficulty=%lx%016lx\n", device_minor_upper, device_minor_lower);
#else
    // printf("n=%lu t=%lu nk=%d bdim=%d bid=%d tid=%d\n", nonce, thread, num_keccak_blocks,
    //        blockDim.x, blockIdx.x, threadIdx.x);

    // if (nonce == 609667058559510631)
    // {
    //     printf("here!!!!\n");
    //     printf("n=%lu t=%lu nk=%d bdim=%d bid=%d tid=%d\n", nonce, thread, num_keccak_blocks,
    //            blockDim.x, blockIdx.x, threadIdx.x);
    // }
#endif

    uint64_t save_state00, save_state01, save_state02, save_state03;

    //absoring phase
    for (int k = 0; k < num_keccak_blocks; k++)
    {

#if 0 < DATA_BLOCK_SIZE
        // state00 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k];
        state00 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+1) << 32);
        // printf("%016llX\n", state00);
#endif

#if 1 < DATA_BLOCK_SIZE
        // state01 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+1];
        state01 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 2) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 3) << 32);

#endif

#if 2 < DATA_BLOCK_SIZE
        // state02 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+2];
        state02 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 4) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 5) << 32);

#endif

#if 3 < DATA_BLOCK_SIZE
        // state03 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+3];
        state03 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 6) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 7) << 32);

#endif

#if 4 < DATA_BLOCK_SIZE
        // state04 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+4];
        state04 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 8) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 9) << 32);

#endif

#if 5 < DATA_BLOCK_SIZE
        // state10 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+5];
        state10 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 10) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 11) << 32);

#endif

#if 6 < DATA_BLOCK_SIZE
        // state11 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+6];
        state11 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 12) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 13) << 32);

#endif

#if 7 < DATA_BLOCK_SIZE
        // state12 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+7];
        state12 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 14) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 15) << 32);

#endif

#if 8 < DATA_BLOCK_SIZE
        // state13 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+8];
        state13 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 16) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 17) << 32);

#endif

#if 9 < DATA_BLOCK_SIZE
        // state14 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+9];
        state14 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 18) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 19) << 32);

#endif

#if 10 < DATA_BLOCK_SIZE
        // state20 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+10];
        state20 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 20) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 21) << 32);

#endif

#if 11 < DATA_BLOCK_SIZE
        // state21 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+11];
        state21 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 22) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 23) << 32);

#endif

#if 12 < DATA_BLOCK_SIZE
        // state22 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+12];
        state22 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 24) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 25) << 32);

#endif

#if 13 < DATA_BLOCK_SIZE
        // state23 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+13];
        state23 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 26) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 27) << 32);

#endif

#if 14 < DATA_BLOCK_SIZE
        // state24 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+14];
        state24 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 28) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 29) << 32);

#endif

#if 15 < DATA_BLOCK_SIZE
        // state30 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+15];
        state30 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 30) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 31) << 32);

#endif

#if 16 < DATA_BLOCK_SIZE
        // state31 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+16];
        state31 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 32) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 33) << 32);

#endif

#if 17 < DATA_BLOCK_SIZE
        // state32 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+17];
        state32 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 34) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 35) << 32);

#endif

#if 18 < DATA_BLOCK_SIZE
        // state33 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+18];
        state33 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 36) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 37) << 32);

#endif

#if 19 < DATA_BLOCK_SIZE
        // state34 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+19];
        state34 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 38) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 39) << 32);

#endif

#if 20 < DATA_BLOCK_SIZE
        // state40 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+20];
        state40 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 40) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 41) << 32);

#endif

#if 21 < DATA_BLOCK_SIZE
        // state41 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+21];
        state41 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 42) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 43) << 32);

#endif

#if 22 < DATA_BLOCK_SIZE
        // state42 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+22];
        state42 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 44) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 45) << 32);

#endif

#if 23 < DATA_BLOCK_SIZE
        // state43 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+23];
        state43 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 46) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 47) << 32);

#endif

#if 24 < DATA_BLOCK_SIZE
        // state44 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+24];
        state44 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 48) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 49) << 32);
#endif

        state03 = cuda_swab64(nonce);

        save_state00 = cuda_swab64(state00);
        save_state01 = cuda_swab64(state01);
        save_state02 = cuda_swab64(state02);
        save_state03 = cuda_swab64(state03);
#if DEBUG
        printf("MSG:\n0x%016lx%016lx%016lx%016lx\n",
               cuda_swab64(state00),
               cuda_swab64(state01),
               cuda_swab64(state02),
               cuda_swab64(state03));
#endif
        // if (nonce == 609667058559510631)
        // {
        //     printf("MSG:\n0x%016lx%016lx%016lx%016lx\n",
        //            cuda_swab64(state00),
        //            cuda_swab64(state01),
        //            cuda_swab64(state02),
        //            cuda_swab64(state03));
        // }

#pragma unroll 4
        for (int i = 0; i < Nr; i++)
        {
            Csum0 = state00 ^ state10 ^ state20 ^ state30 ^ state40;
            Csum1 = state01 ^ state11 ^ state21 ^ state31 ^ state41;
            Csum2 = state02 ^ state12 ^ state22 ^ state32 ^ state42;
            Csum3 = state03 ^ state13 ^ state23 ^ state33 ^ state43;
            Csum4 = state04 ^ state14 ^ state24 ^ state34 ^ state44;
            //
            D0 = Csum4 ^ ROL(Csum1, 1);
            D1 = Csum0 ^ ROL(Csum2, 1);
            D2 = Csum1 ^ ROL(Csum3, 1);
            D3 = Csum2 ^ ROL(Csum4, 1);
            D4 = Csum3 ^ ROL(Csum0, 1);

            state00 ^= D0;
            state01 ^= D1;
            state02 ^= D2;
            state03 ^= D3;
            state04 ^= D4;
            tmpState00 = state00;
            tmpState20 = ROL(state01, 1);
            tmpState40 = ROL(state02, 62);
            tmpState10 = ROL(state03, 28);
            tmpState30 = ROL(state04, 27);

            state10 ^= D0;
            state11 ^= D1;
            state12 ^= D2;
            state13 ^= D3;
            state14 ^= D4;

            tmpState31 = ROL(state10, 36);
            tmpState01 = ROL(state11, 44);
            tmpState21 = ROL(state12, 6);
            tmpState41 = ROL(state13, 55);
            tmpState11 = ROL(state14, 20);

            state20 ^= D0;
            state21 ^= D1;
            state22 ^= D2;
            state23 ^= D3;
            state24 ^= D4;

            tmpState12 = ROL(state20, 3);
            tmpState32 = ROL(state21, 10);
            tmpState02 = ROL(state22, 43);
            tmpState22 = ROL(state23, 25);
            tmpState42 = ROL(state24, 39);

            state30 ^= D0;
            state31 ^= D1;
            state32 ^= D2;
            state33 ^= D3;
            state34 ^= D4;

            tmpState43 = ROL(state30, 41);
            tmpState13 = ROL(state31, 45);
            tmpState33 = ROL(state32, 15);
            tmpState03 = ROL(state33, 21);
            tmpState23 = ROL(state34, 8);

            state40 ^= D0;
            state41 ^= D1;
            state42 ^= D2;
            state43 ^= D3;
            state44 ^= D4;

            //
            tmpState24 = ROL(state40, 18);
            tmpState44 = ROL(state41, 2);
            tmpState14 = ROL(state42, 61);
            tmpState34 = ROL(state43, 56);
            tmpState04 = ROL(state44, 14);

            //
            state00 = tmpState00 ^ ((~tmpState01) & tmpState02);
            state10 = tmpState10 ^ ((~tmpState11) & tmpState12);
            state20 = tmpState20 ^ ((~tmpState21) & tmpState22);
            state30 = tmpState30 ^ ((~tmpState31) & tmpState32);
            state40 = tmpState40 ^ ((~tmpState41) & tmpState42);

            state01 = tmpState01 ^ ((~tmpState02) & tmpState03);
            state11 = tmpState11 ^ ((~tmpState12) & tmpState13);
            state21 = tmpState21 ^ ((~tmpState22) & tmpState23);
            state31 = tmpState31 ^ ((~tmpState32) & tmpState33);
            state41 = tmpState41 ^ ((~tmpState42) & tmpState43);

            state02 = tmpState02 ^ ((~tmpState03) & tmpState04);
            state12 = tmpState12 ^ ((~tmpState13) & tmpState14);
            state22 = tmpState22 ^ ((~tmpState23) & tmpState24);
            state32 = tmpState32 ^ ((~tmpState33) & tmpState34);
            state42 = tmpState42 ^ ((~tmpState43) & tmpState44);

            state03 = tmpState03 ^ ((~tmpState04) & tmpState00);
            state13 = tmpState13 ^ ((~tmpState14) & tmpState10);
            state23 = tmpState23 ^ ((~tmpState24) & tmpState20);
            state33 = tmpState33 ^ ((~tmpState34) & tmpState30);
            state43 = tmpState43 ^ ((~tmpState44) & tmpState40);

            state04 = tmpState04 ^ ((~tmpState00) & tmpState01);
            state14 = tmpState14 ^ ((~tmpState10) & tmpState11);
            state24 = tmpState24 ^ ((~tmpState20) & tmpState21);
            state34 = tmpState34 ^ ((~tmpState30) & tmpState31);
            state44 = tmpState44 ^ ((~tmpState40) & tmpState41);

            state00 ^= RC[i];
        }
    }

    //     //squeezing phase;
    // #if 0 < HASH_SIZE
    //     memcpy(output+(blockIdx.x*BLOCKX + threadIdx.x)*HASH_SIZE, &state00, 8);
    // #endif

    // #if 8 < HASH_SIZE
    //     memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 8, &state01, 8);
    // #endif

    // #if 16 < HASH_SIZE
    //     memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 16, &state02, 8);
    // #endif

    // #if 24 < HASH_SIZE
    //     memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 24, &state03, 8);
    // #endif

#if DEBUG
    // printf("state:0x%016lx\n", cuda_swab64(state00));
    printf("nonce=0x%016lx\nOUT: \n0x%016lx%016lx%016lx%016lx\n",
           nonce,
           cuda_swab64(state00),
           cuda_swab64(state01),
           cuda_swab64(state02),
           cuda_swab64(state03));
#endif

    bool found = 0;
    uint32_t upper = 0;
    uint64_t lower = 0;

    lower = cuda_swab64(state03);
    upper = cuda_swab64(state02);
    upper = upper << 8;

    if (device_difficulty_upper && upper < device_difficulty_upper)
    {
        found = 1;
    }
    else
    {

        if (device_difficulty_upper == upper && lower < device_difficulty_lower)
        {
            found = 1;
        }
    }

    if (found)
    {

        // device_found_nonce = nonce;
        printf("IN: \n0x%016lx%016lx%016lx%016lx\n OUT: \n0x%016lx%016lx%016lx%016lx\n",
               save_state00,
               save_state01,
               save_state02,
               save_state03,
               cuda_swab64(state00),
               cuda_swab64(state01),
               cuda_swab64(state02),
               cuda_swab64(state03));
        printf(">>> FOUND XXX nonce=%lu/0x%016lx combined=0x%06lx%016lx difficulty=0x%06lx%016lx\n", nonce, nonce, upper, lower,
               device_difficulty_upper, device_difficulty_lower);
        device_found_nonce = nonce;
    }
    else if (device_minor_lower)
    {
#if MINOR
        // do same thing for minor nonce
        found = 0;

        if (device_minor_upper && upper < device_minor_upper)
        {
            found = 1;
        }
        else
        {

            if (device_minor_upper == upper && lower < device_minor_lower)
            {
                found = 1;
            }
        }

        if (found)
        {

            printf("IN: \n0x%016lx%016lx%016lx%016lx\n OUT: \n0x%016lx%016lx%016lx%016lx\n",
                   save_state00,
                   save_state01,
                   save_state02,
                   save_state03,
                   cuda_swab64(state00),
                   cuda_swab64(state01),
                   cuda_swab64(state02),
                   cuda_swab64(state03));
            printf(">>> found minor nonce=%lu/0x%016lx combined=0x%06lx%016lx minor=0x%06lx%016lx\n", nonce, nonce, upper, lower,
                   device_minor_upper, device_minor_lower);
            device_found_minor = nonce;
        }
#endif
    }

    atomicAdd(&device_hash_count, 1);

#if DEBUG
    // printf("device_hash_count=%u\n", device_hash_count);
#endif

    // #if 32 < HASH_SIZE
    //     memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 32, &state04, 8);
    // #endif
}

int Padding(uint8_t input[], int inputByte, uint8_t output[])
{
    int outputByte = R / 8 - (inputByte + 1) % (R / 8) + inputByte + 1;
    log_info("Padding inputByte=%d outputByte=%d\n", inputByte, outputByte);
    memcpy(output, input, inputByte);
    memset(output + inputByte, 0, sizeof(uint8_t) * (outputByte - inputByte));
    output[inputByte] = SUFFIX;
    output[outputByte - 1] ^= 0x80;
    return outputByte;
}

//byte

// uint8_t m[] = {0x22, 0x23, 0x3E, 0x5F, 0xCC, 0x4E, 0xFC, 0x0E, 0xEB, 0x03, 0x0C, 0x72, 0xF9, 0x7A, 0x4E, 0x8A, 0x9D, 0xC4, 0xBB, 0x96, 0x18, 0x33, 0xDA, 0xE8, 0xEF, 0xED, 0xCF, 0xFD, 0xE2, 0xA3, 0xC0, 0x37, 0x00, 0x69, 0xCE, 0x65, 0xB3, 0x32, 0x38, 0xAC, 0x43, 0xD6, 0x47, 0x64, 0xFB, 0xDA, 0xDE, 0xDC, 0x6A, 0x22, 0xA3, 0x0C, 0x15, 0xCC, 0x01, 0x0D, 0x7F, 0xC3, 0xA4, 0x45, 0xE3, 0x5E, 0xDA, 0xB7, 0x69, 0x29, 0xD0, 0xAB, 0x6C, 0x48, 0x35, 0xF2, 0x1F, 0xA7, 0x2D, 0x20, 0xC3, 0x3E, 0x5F, 0xCC, 0x4E, 0xFC, 0x0E, 0xEB, 0x03, 0x0C, 0x72, 0xF9, 0x7A, 0x4E, 0x8A, 0x9D, 0xC4, 0xBB, 0x96, 0x18, 0x33, 0xDA, 0xE8, 0xEF, 0xED, 0xCF, 0xFD, 0xE2, 0xA3, 0xC0, 0x37, 0x00, 0x69, 0xCE, 0x65, 0xB3, 0x32, 0x38, 0xAC, 0x43, 0xD6, 0x47, 0x64, 0xFB, 0xDA, 0xDE, 0xDC};
// uint8_t msg[32] = {0x04, 0x22, 0x00, 0x00, 0x00, 0x00, 0x3B, 0x00, 0x19, 0x00, 0x00, 0x00,
//                  0x7D, 0x43, 0x7E, 0x28, 0xCD, 0x73, 0xA3, 0xF4, 0x87,
//                  0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};
uint8_t msg[32] = {0};

uint8_t output[BLOCKNUM * BLOCKX][HASH_SIZE];
uint8_t input[BLOCKSIZE];
uint8_t host_input[SUMDATASIZE];

// #define STREAMNUM 5 xxx

hipStream_t stream[STREAMNUM];
uint32_t *device_input[STREAMNUM];
uint8_t *device_output[STREAMNUM];

uint64_t getTime(void)
{
    uint64_t val = 0;
    struct timeval tv;

    gettimeofday(&tv, NULL);

    val = (((uint64_t)tv.tv_sec) * 1000 + ((uint64_t)tv.tv_usec) / 1000);
    // log_info("getTime tv.tv_sec %ld tv_usec %ld val %ld\n", tv.tv_sec, tv.tv_usec, val);
    return (uint64_t)val;
}

void printMsg(const char *title, uint8_t *msg, int len)
{
    if (title)
    {
        log_info("%s:\n0x", title);
    }
    else
    {
        printf("0x");
    }
    for (int i = 0; i < len; i++)
    {
        printf("%02X", msg[i]);
    }
    printf("\n");
}

void FreeAll()
{
    log_info("freeAll..\n");
    hipDeviceSynchronize();
    for (int i = 0; i < STREAMNUM; i++)
    {
        hipStreamDestroy(stream[i]);
        hipFree(device_input[i]);
        hipFree(device_output[i]);
    }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        log_err("Cuda error : % s : % s.\n ", msg, hipGetErrorString(err));
        FreeAll();
        exit(EXIT_FAILURE);
    }
}

void setMsg(OPTS *opts)
{

    const char *val;
    int base;
    mpz_t sender_mpz;
    mpz_t lastMinedPunkAsset_mpz;
    mpz_t difficulty_mpz;
    mpz_t startNonce_mpz;
    size_t count;

    if (opts->str_address)
    {
        val = opts->str_address;
    }
    else
    {
        val = DEFAULT_ADDRESS;
    }
    if (val && val[0] == '0' and val[1] == 'x')
    {
        val = val + 2;
        base = 16;
    }
    else
    {
        base = 10;
    }
    mpz_init_set_str(sender_mpz, &val[22], 16);
    gmp_printf("sender_mpz=%Zd/%018Zx\n", sender_mpz, sender_mpz);

    if (opts->str_lastMined)
    {
        val = opts->str_lastMined;
    }
    else
    {
        val = DEFAULT_LASTMINED;
    }
    if (val && val[0] == '0' and val[1] == 'x')
    {
        val = val + 2;
        base = 16;
    }
    else
    {
        base = 10;
    }

    mpz_init_set_str(lastMinedPunkAsset_mpz, val, base);
    gmp_printf("lastMinedPunkAsset_mpz=%Zd/0x%Zx\n", lastMinedPunkAsset_mpz, lastMinedPunkAsset_mpz);

    if (opts->str_startNonce)
    {
        val = opts->str_startNonce;
    }
    else
    {
        val = NULL;
    }

    if (val && val[0] == '0' and val[1] == 'x')
    {
        val = val + 2;
        base = 16;
    }
    else
    {
        base = 10;
    }

    if (val)
    {
        mpz_init_set_str(startNonce_mpz, val, base);
        gmp_printf("startNonce_mpz=%Zd/0x%Zx\n", startNonce_mpz, startNonce_mpz);
        mpz_export(&opts->startNonce, &count, 1, sizeof(opts->startNonce), 0, 0, startNonce_mpz);
    }

    if (opts->str_difficulty)
    {
        val = opts->str_difficulty;
    }
    else
    {
        val = DEFAULT_DIFFICULTY;
    }

    if (val && val[0] == '0' and val[1] == 'x')
    {
        val = val + 2;
        base = 16;
    }
    else
    {
        base = 10;
    }

    uint8_t difficulty[16];
    if (val)
    {
        mpz_init_set_str(difficulty_mpz, val, base);
        gmp_printf("difficulty_mpz=%Zd/0x%032Zx\n", difficulty_mpz, difficulty_mpz);
        mpz_export(difficulty, &count, 1, sizeof(difficulty), 0, 0, difficulty_mpz);

        opts->upper_difficulty = ((uint64_t *)difficulty)[1];
        opts->lower_difficulty = ((uint64_t *)difficulty)[0];
    }
    // printMsg("difficulty", difficulty, 16);

    // log_info("0x%016lx %016lx\n", opts->upper_difficulty, opts->lower_difficulty);

    if (opts->str_minor)
    {
        val = opts->str_minor;
    }
    else
    {
        val = DEFAULT_MINOR;
    }

    if (val && val[0] == '0' and val[1] == 'x')
    {
        val = val + 2;
        base = 16;
    }
    else
    {
        base = 10;
    }

    if (val)
    {
        mpz_init_set_str(difficulty_mpz, val, base);
        gmp_printf("minor difficulty_mpz=%Zd/0x%032Zx\n", difficulty_mpz, difficulty_mpz);
        mpz_export(difficulty, &count, 1, sizeof(difficulty), 0, 0, difficulty_mpz);

        opts->upper_minor = ((uint64_t *)difficulty)[1];
        opts->lower_minor = ((uint64_t *)difficulty)[0];
    }
    // printMsg("difficulty", difficulty, 16);

    /* set msg */
    printMsg("pre msg", msg, 32);
    mpz_export(msg, &count, 1, 12, 1, 0, lastMinedPunkAsset_mpz);
    mpz_export(msg + 12, &count, 1, 9, 1, 0, sender_mpz);
    printMsg("pos msg", msg, 32);

    Padding(msg, sizeof(msg), input);
    for (int i = 0; i < STREAMNUM; i++)
    {
        hipStreamCreate(&stream[i]);
    }
    checkCUDAError("create stream error");
    log_info("init.. writing %d blocks size_t=%d\n", BLOCKX * BLOCKNUM, BLOCKSIZE);
    for (int i = 0; i < BLOCKX * BLOCKNUM; i++)
    {
        memcpy(host_input + i * BLOCKSIZE, input, BLOCKSIZE);
        // printMsg("msg",host_input + i*BLOCKSIZE, 32);

        // break;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(device_difficulty_lower), &opts->lower_difficulty, sizeof(opts->lower_difficulty), 0, hipMemcpyHostToDevice);
    checkCUDAError("copy to symbol");
    hipMemcpyToSymbol(HIP_SYMBOL(device_difficulty_upper), &opts->upper_difficulty, sizeof(opts->upper_difficulty), 0, hipMemcpyHostToDevice);
    checkCUDAError("copy to symbol");

    hipMemcpyToSymbol(HIP_SYMBOL(device_minor_lower), &opts->lower_minor, sizeof(opts->lower_minor), 0, hipMemcpyHostToDevice);
    checkCUDAError("copy to symbol");
    hipMemcpyToSymbol(HIP_SYMBOL(device_minor_upper), &opts->upper_minor, sizeof(opts->upper_minor), 0, hipMemcpyHostToDevice);
    checkCUDAError("copy to symbol");
}

void GetCudaMalloc(int length)
{
    for (int i = 0; i < STREAMNUM; i++)
    {
        hipMalloc(&device_input[i], BLOCKNUM * BLOCKX * BLOCKSIZE);
        checkCUDAError("malloc for device_input");
        hipMalloc(&device_output[i], BLOCKX * BLOCKNUM * HASH_SIZE);
        checkCUDAError("malloc for device_output");
    }
}

static int destructing = 0;
void destruct()
{
    log_info("destruct..\n");
    if (destructing)
    {
        return;
    }
    destructing = 1;
}

/* Signal Handler for SIGINT */
void sigintHandler(int sig_num)
{
    log_info("caught signal: SIGINT\n");

    destruct();
}

/* Signal Handler for SIGTERM */
void sigtermHandler(int sig_num)
{
    log_info("caught signal: SIGTERM\n");
    destruct();
}

void get_options(int argc, char **argv, OPTS *opts)
{
    int c;

    memset(opts, 0, sizeof(OPTS));

    opts->controller = DEFAULT_CONTROLLER;
    opts->str_address = strdup(DEFAULT_ADDRESS);
    opts->start_address = strdup(DEFAULT_ADDRESS);
    opts->str_difficulty = strdup(DEFAULT_DIFFICULTY);
    opts->str_lastMined = strdup(DEFAULT_LASTMINED);

    static struct option long_options[] =
        {
            {"address", required_argument, 0, 'a'},
            {"difficulty", required_argument, 0, 'd'},
            {"startNonce", required_argument, 0, 's'},
            {"lastMined", required_argument, 0, 'l'},
            {"cudaDevice", required_argument, 0, 'x'},
            {"testing", no_argument, 0, 't'},
            {"user controller flag", optional_argument, 0, 'c'},
            {"version", no_argument, 0, 'v'},
            {"help", no_argument, 0, 'h'},
            {0, 0, 0, 0}};

#if FULL
#else
    opts->use_controller = true;
#endif

    while (1)
    {
        int option_index = 0;

        c = getopt_long(argc, argv, "a:d:s:l:x:tc::vh", long_options, &option_index);

        /* Detect the end of the options. */
        if (c == -1)
            break;

        switch (c)
        {
        case '0':
            log_info("have 0\n");
            break;
        case 'a':
#if FULL
#else
            free(opts->str_address);
            opts->str_address = strdup(optarg);
#endif
            free(opts->start_address);
            opts->start_address = strdup(optarg);
            log_info("opt address='%s'\n", opts->str_address);
            break;
        case 'd':
            free(opts->str_difficulty);
            opts->str_difficulty = strdup(optarg);
            log_info("opt difficulty='%s'\n", opts->str_difficulty);
            break;
        case 's':
            free(opts->str_startNonce);
            opts->str_startNonce = strdup(optarg);
            log_info("opt startNonce='%s'\n", opts->str_startNonce);
            break;
        case 'l':
            free(opts->str_lastMined);
            opts->str_lastMined = strdup(optarg);
            log_info("opt lastMined='%s'\n", opts->str_lastMined);
            break;
        case 'x':
            opts->device = atoi(optarg);
            log_info("opt device='%d'\n", opts->device);
            break;
        case 't':
            opts->test = true;
            log_info("opt test only\n");
            break;
        case 'c':
            opts->use_controller = true;
#if FULL
            if (optarg) // XXX
            {
                opts->controller = strdup(optarg);
            }
            log_info("use controller=%s\n", opts->controller);
#endif
            break;
        case 'v':
            printf("version=%s\n", VERSION);
            exit(0);
        default:
            log_info("option `%c` is not supported.\n", c);
            exit(0);
        }
    }
}

struct MemoryStruct
{
    char *memory;
    size_t size;
};

size_t
getCurlData(void *contents, size_t size, size_t nmemb, void *userp)
{
    size_t realsize = size * nmemb;
    struct MemoryStruct *mem = (struct MemoryStruct *)userp;
    mem->memory = (char *)realloc(mem->memory, mem->size + realsize + 1);
    if (mem->memory == NULL)
    {
        /* out of memory! */
        log_err("not enough memory (realloc returned NULL)\n");
        return 0;
    }
    memcpy(&(mem->memory[mem->size]), contents, realsize);
    mem->size += realsize;
    mem->memory[mem->size] = 0;
    return realsize;
}

bool json_setValue(char **place, json_t *payload, const char *name, bool *changed)
{

    json_t *value = json_object_get(payload, name);
    if (!value)
    {
        log_info("error unable to get %s.\n", name);
        return false;
    }

    json_auto_t *compare = json_string(*place);

    if (!json_equal(compare, value))
    {
        *changed = true;
    }

    // log_info("%p\n", *place);

    if (*changed)
    {
        free(*place);
        *place = strdup((char *)json_string_value(value));
        log_info("controller setting %s=%s\n", name, *place);
    }

    json_decref(compare);

    return true;
}

bool submitNonce(OPTS *opts, uint64_t nonce, bool minor)
{

    if (destructing)
        return false;

    CURL *curl;
    CURLcode res;
    // bool success = false;

    log_info("submitNonce.. nonce=%lx\n", nonce);

    curl = curl_easy_init();

    struct MemoryStruct chunk;
    chunk.memory = NULL;
    chunk.size = 0;
    chunk.memory = (char *)malloc(1);

    char url[256];
    const char *address = opts->str_address;
    if (!address)
    {
        address = DEFAULT_ADDRESS;
    }

    if (minor == true)
    {
        sprintf(url, "%s/submit-ping?nonce=%lu&address=%s&last=%s&src=%s", opts->controller, nonce, address, opts->str_lastMined,
                opts->start_address);
    }
    else
    {
        sprintf(url, "%s/submit-work?nonce=%lu&address=%s&last=%s&src=%s", opts->controller, nonce, address, opts->str_lastMined,
                opts->start_address);
    }
    log_sensitive("url=%s\n", url);

    curl_easy_setopt(curl, CURLOPT_URL, url);
    curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, getCurlData);
    curl_easy_setopt(curl, CURLOPT_WRITEDATA, (void *)&chunk);
    res = curl_easy_perform(curl);
    if (res != CURLE_OK)
    {
        log_err("curl_easy_perform() failed: %s\n", curl_easy_strerror(res));
        if (chunk.memory)
        {
            free(chunk.memory);
        }
        curl_easy_cleanup(curl);
        return false;
    }

    if (!chunk.memory)
    {
        log_info("chunk memory is null\n");
        curl_easy_cleanup(curl);
        return false;
    }

    log_info("response: %s\n", chunk.memory);
    free(chunk.memory);
    curl_easy_cleanup(curl);
    return true;
}

bool submitMinor(OPTS *opts, uint64_t nonce)
{
    log_info("submitMinor..\n");
    return submitNonce(opts, nonce, true);
}

bool heartbeat(OPTS *opts, uint32_t hash_rate)
{
    if (destructing)
        return false;

    CURL *curl;
    CURLcode res;
    // bool success = false;

    log_info("heartbeat.. hash_rate=%u\n", hash_rate);

    curl = curl_easy_init();

    struct MemoryStruct chunk;
    chunk.memory = NULL;
    chunk.size = 0;
    chunk.memory = (char *)malloc(1);

    char url[256];
    const char *address = opts->str_address;
    if (!address)
    {
        address = DEFAULT_ADDRESS;
    }
    sprintf(url, "%s/heartbeat?hashrate=%u&address=%s&src=%s", opts->controller, hash_rate, address, opts->start_address);

    log_sensitive("url=%s\n", url);

    curl_easy_setopt(curl, CURLOPT_URL, url);
    curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, getCurlData);
    curl_easy_setopt(curl, CURLOPT_WRITEDATA, (void *)&chunk);
    res = curl_easy_perform(curl);
    if (res != CURLE_OK)
    {
        log_err("curl_easy_perform() failed: %s\n", curl_easy_strerror(res));
        if (chunk.memory)
        {
            free(chunk.memory);
        }
        curl_easy_cleanup(curl);
        return false;
    }

    if (!chunk.memory)
    {
        log_info("chunk memory is null\n");
        curl_easy_cleanup(curl);
        return false;
    }

    free(chunk.memory);
    curl_easy_cleanup(curl);
    return true;
}

bool getMiningInputs(OPTS *opts)
{

    if (destructing)
        return false;

    CURL *curl;
    CURLcode res;
    bool success = false;

    log_info("getMiningInputs..\n");

    curl = curl_easy_init();

    struct MemoryStruct chunk;
    chunk.memory = NULL;
    chunk.size = 0;
    chunk.memory = (char *)malloc(1);

    char url[256];
    const char *address = opts->str_address;
    if (!address)
    {
        address = DEFAULT_ADDRESS;
    }
    sprintf(url, "%s/mining-inputs?address=%s", opts->controller, address);

    log_sensitive("url=%s\n", url);

    curl_easy_setopt(curl, CURLOPT_URL, url);
    curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, getCurlData);
    curl_easy_setopt(curl, CURLOPT_WRITEDATA, (void *)&chunk);
    res = curl_easy_perform(curl);
    if (res != CURLE_OK)
    {
        log_err("curl_easy_perform() failed: %s\n", curl_easy_strerror(res));
        if (chunk.memory)
        {
            free(chunk.memory);
        }
        return false;
    }

    if (!chunk.memory)
    {
        log_info("chunk memory is null\n");
        curl_easy_cleanup(curl);
        return false;
    }

    json_error_t error;
    json_t *root;

    root = json_loads(chunk.memory, 0, &error);
    if (!root)
    {
        log_info("error loading json %s\n", error.text);
        log_info("data %s\n", chunk.memory);
        if (chunk.memory)
        {
            free(chunk.memory);
        }
        curl_easy_cleanup(curl);
        return false;
    }
    else
    {
        json_t *value = json_object_get(root, "status");
        json_auto_t *compare = json_string("success");
        if (!json_equal(value, compare))
        {
            log_info("not successful %s\n", chunk.memory);
            json_decref(root);
            json_decref(compare);
            goto end;
        }
        json_decref(compare);

        json_t *payload = json_object_get(root, "payload");
        if (!payload)
        {
            log_info("unable to get payload. %s\n", chunk.memory);
            json_decref(root);
            goto end;
        }

        bool changed = 0;
        success = json_setValue(&opts->str_lastMined, payload, "lastMinedAssets", &changed);
        if (!success)
        {
            log_info("error data: %s\n", chunk.memory);
            json_decref(root);
            goto end;
        }

        success = json_setValue(&opts->str_address, payload, "senderAddress", &changed);
        if (!success)
        {
            log_info("error data: %s\n", chunk.memory);
            json_decref(root);
            goto end;
        }

        success = json_setValue(&opts->str_difficulty, payload, "difficultyTarget", &changed);
        if (!success)
        {
            log_info("error data: %s\n", chunk.memory);
            json_decref(root);
            goto end;
        }

        success = json_setValue(&opts->str_minor, payload, "minorDifficulty", &changed);
        if (!success)
        {
            log_info("error data: %s\n", chunk.memory);
            json_decref(root);
            goto end;
        }

        opts->values_changed = changed;

        json_decref(root);
    }
end:
    if (chunk.memory)
    {
        free(chunk.memory);
    }
    curl_easy_cleanup(curl);
    return success;
}

int main(int argc, char **argv)
{
    log_info("Hi There!!\n");

    /* xxx random number */
    time_t t;
    srand((unsigned)time(&t) + (unsigned)getpid());

    signal(SIGINT, sigintHandler);
    signal(SIGTERM, sigtermHandler);

    OPTS opts;
    get_options(argc, argv, &opts);

    if (opts.use_controller)
    {
        getMiningInputs(&opts);
    }

    log_info("using device %d\n.", opts.device);
    hipSetDevice(opts.device);
    checkCUDAError("set device");

    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, Keccak1600, BLOCKSIZE, 0);
    log_info("recomminding blockSize=%d gridSize=%d\n", minGridSize, blockSize);

    // opts.block_size = blockSize;
    // opts.grid_size = minGridSize;

    if (opts.test)
    {
        return 0;
    }
    GetCudaMalloc(BLOCKSIZE);

    timeval tpstart;

    log_info("CUDA start\n");
    int cur = 0;
    gettimeofday(&tpstart, NULL);
    // double all_sec = 0;
    uint64_t start = getTime();
    uint64_t tstart = start;
    uint64_t elapsed = 0;
    uint32_t n_hashes = 0;
    uint32_t hash_count = 0;
    uint32_t hash_rate = 0;

    uint64_t found_nonce = 0;
    uint64_t found_minor = 0;

    int n_secs = 0;

    hipEvent_t cuda_start, cuda_stop;

    setMsg(&opts);
    for (int i = 0; i < STREAMNUM; i++)
    {
        hipMemcpyAsync(device_input[i], host_input, SUMDATASIZE, hipMemcpyHostToDevice, stream[i]);
        checkCUDAError("memcpy from buf to device_input");
    }

    uint64_t startNonce;
    int run = 0;
#if DEBUG
    if (opts.str_startNonce)
    {
        startNonce = opts.startNonce;
    }
    else
    {
        startNonce = 609667058559510624;
    }

    for (int i = 0; i < 3; i++)
#else
    if (opts.str_startNonce)
    {
        startNonce = opts.startNonce;
    }
    else
    {
        startNonce = rand_uint64();
    }
    // startNonce = 609667058559510630;
    while (!destructing)
    // for (int i = 0; i < 2; i++)
#endif
    {

#if DEBUG
        log_info("%s run=%d startNonce=%lu/0x%016lx ->>\n", ctime(&t), run, startNonce, startNonce);
#endif

        hipBindTexture(0, texreference_input, device_input[cur], SUMDATASIZE);

        hipEventCreate(&cuda_start);
        hipEventCreate(&cuda_stop);
        hipEventRecord(cuda_start, 0);

        Keccak1600<<<BLOCKNUM, BLOCKX, 0, stream[cur]>>>(BLOCKSIZE / 4, device_output[cur], HASH_SIZE, startNonce);

        hipEventRecord(cuda_stop, 0);
        hipEventSynchronize(cuda_stop);

        float elapsedTime = 0.0;
        hipEventElapsedTime(&elapsedTime, cuda_start, cuda_stop);
        hipMemcpyFromSymbol(&hash_count, HIP_SYMBOL(device_hash_count), sizeof(hash_count), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&found_nonce, HIP_SYMBOL(device_found_nonce), sizeof(found_nonce), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&found_minor, HIP_SYMBOL(device_found_minor), sizeof(found_minor), 0, hipMemcpyDeviceToHost);
        // log_info("device took %fms for %u hashes\n", elapsedTime, hash_count);

        hipEventDestroy(cuda_start);
        hipEventDestroy(cuda_stop);

        cur = (cur + 1) % STREAMNUM;
        hipUnbindTexture(&texreference_input);

        // log_info("hash_count=%d\n", hash_count);

        if (found_nonce)
        {
            log_info(">>>>>>>>>>>found_nonce=%lu\n", found_nonce);
            submitNonce(&opts, found_nonce, false);
            found_nonce = 0;
            hipMemcpyToSymbol(HIP_SYMBOL(device_found_nonce), &found_nonce, sizeof(found_nonce), 0, hipMemcpyHostToDevice);
        }

        if (found_minor)
        {
            log_info(">>>>>>>>>>>found_minor=%lu\n", found_minor);
            submitMinor(&opts, found_minor);
            found_minor = 0;
            hipMemcpyToSymbol(HIP_SYMBOL(device_found_minor), &found_minor, sizeof(found_minor), 0, hipMemcpyHostToDevice);
        }

        // hash_count = BLOCKX * BLOCKNUM;

        startNonce += hash_count;
        n_hashes += hash_count;
        hash_count = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(device_hash_count), &hash_count, sizeof(hash_count), 0, hipMemcpyHostToDevice);

        elapsed = getTime() - tstart;
        if (elapsed > 1000)
        {
            hash_rate = (n_hashes / elapsed) * 1000;
            log_info(">>> STATS.. nhashes=%u/s n_secs=%ds nonce=%lu\n", hash_rate, n_secs, startNonce);
            n_hashes = 0;
            tstart = getTime();
            n_secs++;
        }

        if (n_secs > POLL_TIME && !destructing && opts.use_controller)
        {
            heartbeat(&opts, hash_rate);
            bool success = getMiningInputs(&opts);
            if (opts.values_changed)
            {
                setMsg(&opts);
                for (int i = 0; i < STREAMNUM; i++)
                {
                    hipMemcpyAsync(device_input[i], host_input, SUMDATASIZE, hipMemcpyHostToDevice, stream[i]);
                    checkCUDAError("memcpy from buf to device_input");
                }

                opts.values_changed = 0;
            }
            n_secs = 0;
        }

        run++;
        fflush(stdout);
    }
    FreeAll();
    log_info("END\n");

    return 0;
}
